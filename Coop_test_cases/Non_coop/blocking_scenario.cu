#include "hip/hip_runtime.h"
/*
Copyright (c) 2015-present Advanced Micro Devices, Inc. All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#define CHECK(cmd) \
{\
    hipError_t error  = cmd;\
    if (error != hipSuccess) { \
        fprintf(stderr, "error: '%s'(%d) at %s:%d\n", hipGetErrorString(error), error,__FILE__, __LINE__); \
        exit(EXIT_FAILURE);\
	  }\
}

/* 
 * Square each element in the array A and write to array C.
 */
template <typename T>
__global__ void
vector_square(T *C_d, T *A_d, size_t N)
{

    size_t offset = (blockIdx.x * blockDim.x + threadIdx.x);
    size_t stride = blockDim.x * gridDim.x ;
//printf("offset: %d\n",offset);
    for (size_t i=offset; i<N; i+=stride) {
        C_d[i] = A_d[i] * A_d[i];
    }
// __SYNC_ACROSS_BLOCKS

if(offset == 0)
printf("I am from thread 0\n");
else if(offset == 40)
printf("I am from thread 40 \n");
__syncthreads();
if(offset==40){
//__syncthreads();
unsigned long long int wait_t=3200000000,start=clock64(),cur;
    do{cur=clock64()-start;}
    while(cur<wait_t);
printf("Wait is over!\n");
}
if(offset == 0)
printf("I am after grid.sync() from thread 0\n");
else if(offset == 40)
printf("I am after grid.sync() from thread 40 \n");


/*// BLOCKING_SCENARIO
if(offset < 16){
__syncthreads();
}
__syncthreads();
*/

}


int main(int argc, char *argv[])
{
    CHECK(hipSetDevice(2));
    float *A_d, *C_d;
    float *A_h, *C_h;
    //size_t N = 1000000;
    size_t N = 64;
    size_t Nbytes = N * sizeof(float);

    hipDeviceProp_t props;
    CHECK(hipGetDeviceProperties(&props, 0/*deviceID*/));
    printf ("info: running on device %s\n", props.name);

    printf ("info: allocate host mem (%6.2f MB)\n", 2*Nbytes/1024.0/1024.0);
    A_h = (float*)malloc(Nbytes);
    CHECK(A_h == 0 ? hipErrorOutOfMemory : hipSuccess );
    C_h = (float*)malloc(Nbytes);
    CHECK(C_h == 0 ? hipErrorOutOfMemory : hipSuccess );
    // Fill with Phi + i
    for (size_t i=0; i<N; i++) 
    {
        A_h[i] = 1.618f + i; 
    }

    printf ("info: allocate device mem (%6.2f MB)\n", 2*Nbytes/1024.0/1024.0);
    CHECK(hipMalloc(&A_d, Nbytes));
    CHECK(hipMalloc(&C_d, Nbytes));


    printf ("info: copy Host2Device\n");
    CHECK ( hipMemcpy(A_d, A_h, Nbytes, hipMemcpyHostToDevice));

    //const unsigned blocks = 512;
    //const unsigned threadsPerBlock = 256;
    //const unsigned blocks = (N+31)/32;
    //const unsigned blocks = 64;
    const unsigned threadsPerBlock = 32;
    const unsigned blocks = N/threadsPerBlock;

    printf ("info: launch 'vector_square' kernel\n");
    vector_square <<<blocks, threadsPerBlock>>> (C_d, A_d, N);
    printf ("info: copy Device2Host\n");
    CHECK ( hipMemcpy(C_h, C_d, Nbytes, hipMemcpyDeviceToHost));

    printf ("info: check result\n");
    for (size_t i=0; i<N; i++)  {
        if (C_h[i] != A_h[i] * A_h[i]) {
            CHECK(hipErrorUnknown);
        }
    }
    printf ("PASSED!\n");
}
